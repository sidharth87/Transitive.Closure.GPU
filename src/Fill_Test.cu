#include "Matrix_Test.h"
#include "load_matrix.h"

void FillTests(const std::string &filename);
void Matrix_Test(const std::string filename);
void createStreams();
void createStreams(const int ID);
void destroyStreams();
void destroyStreams(const int ID);

#include "Fill_Test.inl"

void createStreams(const int ID)
{
	for(int i=0; i<NUM_STREAMS; i++)
		hipStreamCreate(&__multiStreams[ID][i]);
}

void destroyStreams(const int ID)
{
	for(int i=0; i<NUM_STREAMS; i++)
		hipStreamDestroy(__multiStreams[ID][i]);
}

void createStreams()
{
	for(int i=0; i<NUM_STREAMS; i++)
		hipStreamCreate(&__streams[i]);
}

void destroyStreams()
{
	for(int i=0; i<NUM_STREAMS; i++)
		hipStreamDestroy(__streams[i]);
}

void Matrix_Test(const std::string filename)
{
	#if(MULTI_GPU == 1)
		//FillTests(filename);
	#else
		FillTests(filename);
	#endif
}

////////////////////////////////////////////////////////////////////////////////
//	Parse input file and run test
////////////////////////////////////////////////////////////////////////////////

void runTest(int argc, char** argv)
{
	if(argc != 2)
	{
		fprintf(stderr, "Invalid input...\n");
		fprintf(stderr, "Usage: CFA <testfile>\n");
		exit(1);
	}

	std::string filename(argv[1]);
	Matrix_Test(filename);
}

int main(int argc, char **argv)
{
	fprintf(stderr, "TEST START\n");
	runTest(argc, argv);
	fprintf(stderr, "TEST COMPLETE\n");
	return 0;
}