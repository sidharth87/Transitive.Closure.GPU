#include "Matrix_Test.h"
#include "load_matrix.h"
#include "dynamic_buffer.h"
#include "spine.h"

void Transitive_closure(const std::string &filename);
void Matrix_Test(const std::string filename);
void createStreams();
void createStreams(const int ID);
void destroyStreams();
void destroyStreams(const int ID);

#include "Transitive_Closure.inl"

void createStreams(const int ID)
{
	for(int i=0; i<NUM_STREAMS; i++)
		hipStreamCreate(&__multiStreams[ID][i]);
}

void destroyStreams(const int ID)
{
	for(int i=0; i<NUM_STREAMS; i++)
		hipStreamDestroy(__multiStreams[ID][i]);
}

void createStreams()
{
	for(int i=0; i<NUM_STREAMS; i++)
		hipStreamCreate(&__streams[i]);
}

void destroyStreams()
{
	for(int i=0; i<NUM_STREAMS; i++)
		hipStreamDestroy(__streams[i]);
}

void Matrix_Test(const std::string filename)
{
	#if(MULTI_GPU == 1)
		//FillTests(filename);
	#else
		Transitive_closure(filename);
	#endif
}

////////////////////////////////////////////////////////////////////////////////
//	Parse input file and run test
////////////////////////////////////////////////////////////////////////////////

void runTest(int argc, char** argv)
{
	if(argc != 2)
	{
		fprintf(stderr, "Invalid input...\n");
		fprintf(stderr, "Usage: CFA <testfile>\n");
		exit(1);
	}

	std::string filename(argv[1]);
	Transitive_closure(filename);
}

int main(int argc, char **argv)
{
	fprintf(stderr, "TEST START\n");
	runTest(argc, argv);
	fprintf(stderr, "TEST COMPLETE\n");
	return 0;
}
